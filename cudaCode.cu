#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdio.h>
#include "opencv2/opencv.hpp"
using namespace cv;
using namespace std;

/*
    Compile and run instructions: Do not use nvcc. Use the two commands below to compile.
    cmake .
    make
    
    To run the program =>   ./cudaProj
*/

#define BLUR_SIZE 32

__global__ void rgb2grayincuda(uchar3 * const d_in, unsigned char * const d_out, 
                                uint imgheight, uint imgwidth)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < imgwidth && idy < imgheight)
    {
        uchar3 rgb = d_in[idy * imgwidth + idx];
        d_out[idy * imgwidth + idx] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
    }
}

// Implement blur kernel using the book code
__global__ void blurKernel(uchar3 * const d_in, uchar3 * const d_out, 
                                uint imgheight, uint imgwidth)
{
    /* 
        Chapter 3, Section 3.3 contains the code. The chapter pdf is present in Class's D2L Contents under GPU
        The difference between book code and the code here is the data structure that stores
        the pixel values. uchar3 is a structure that contains Red, Green and Blue values. 
        Book code contains one value per pixel.
    */
	const unsigned int Col = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int Row = blockIdx.y * blockDim.y + threadIdx.y;

	if (Col < imgheight && Row < imgwidth){
		int pixValx = 0;
		int pixValy = 0;
		int pixValz = 0;
		int pixels = 0;

		//Get the average of the surrounding BlUR_SIZE x BLUR_SIZE box
		for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow){
			for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol){
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				
				//Verify we have a valid image pixel
				if(curRow > -1 && curRow < imgheight && curCol > -1 && curCol < imgwidth){
					pixValx += d_in[curRow * imgwidth + curCol].x;
					pixValy += d_in[curRow * imgwidth + curCol].y;
					pixValz += d_in[curRow * imgwidth + curCol].z;
					pixels++; //Keep track of number of pixels in avg
				}
			}
		}
		//Write our new pixel value out
		d_out[Row * imgwidth + Col].x = (unsigned char)(pixValx / pixels);
		d_out[Row * imgwidth + Col].y = (unsigned char)(pixValy / pixels);
		d_out[Row * imgwidth + Col].z = (unsigned char)(pixValz / pixels);
	}
}

int main(void)
{
    Mat srcImage = imread("./e1.jpg");
    const uint imgheight = srcImage.rows;
    const uint imgwidth = srcImage.cols;

    Mat grayImage(imgheight, imgwidth, CV_8UC1, Scalar(0));

    uchar3 *d_in;
    unsigned char *d_out;

    hipMalloc((void**)&d_in, imgheight*imgwidth*sizeof(uchar3));
    hipMalloc((void**)&d_out, imgheight*imgwidth*sizeof(unsigned char));

    hipMemcpy(d_in, srcImage.data, imgheight*imgwidth*sizeof(uchar3), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    rgb2grayincuda<< <blocksPerGrid, threadsPerBlock>> >(d_in, d_out, imgheight, imgwidth);

    hipMemcpy(grayImage.data, d_out, imgheight*imgwidth*sizeof(unsigned char), hipMemcpyDeviceToHost);
	
    hipFree(d_out);

	imwrite("greyImage.jpg",grayImage);
	
	
	// For Image blur
	Mat blurImage(imgheight, imgwidth, CV_8UC3);
	uchar3 *d_out2;
	hipMalloc((void**)&d_out2, imgheight*imgwidth*sizeof(uchar3));
	blurKernel<< <blocksPerGrid, threadsPerBlock>> >(d_in, d_out2, imgheight, imgwidth);
	hipMemcpy(blurImage.data, d_out2, imgheight*imgwidth*sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out2);
	imwrite("blurImage32.jpg",blurImage);
	
    return 0;

}
